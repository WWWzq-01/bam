#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <nvm_ctrl.h>
#include <nvm_types.h>
#include <nvm_queue.h>
#include <nvm_util.h>
#include <nvm_admin.h>
#include <nvm_error.h>
#include <nvm_cmd.h>
#include <string>
#include <stdexcept>
#include <vector>
#include <cstdio>
#include <cstdint>
#include <cstring>
#include <fcntl.h>
#include <unistd.h>
#include <sys/mman.h>
#include <ctrl.h>
#include <buffer.h>
#include "settings.h"
#include <event.h>
#include <queue.h>
#include <nvm_parallel_queue.h>
#include <nvm_io.h>
#include <page_cache.h>
#include <util.h>
#include <iostream>
#include <fstream>
#ifdef __DIS_CLUSTER__
#include <sisci_api.h>
#endif

using error = std::runtime_error;
using std::string;

uint32_t n_ctrls = 1;
const char* const ctrls_paths[] = {"/dev/libnvm0", "/dev/libnvm1", "/dev/libnvm2", "/dev/libnvm3", "/dev/libnvm4", "/dev/libnvm5", "/dev/libnvm6", "/dev/libnvm7"};


/*template<typename T>
__global__ __launch_bounds__(64,32)
void random_access_warp(array_d_t<T>* dr, uint64_t n_pages_per_warp, unsigned long long* sum, uint64_t type, uint64_t* assignment, uint64_t n_warps, size_t page_size, size_t sector_size) {

    const uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    const uint64_t lane = tid % 32;
    const uint64_t warp_id = tid / 32;
    const uint64_t n_elems_per_page = page_size/sizeof(T);
    const uint64_t n_elems_per_sector = sector_size / sizeof(T);
    T v = 0;
    if (warp_id < n_warps) {
    	bam_ptr<T> ptr(dr);
        size_t start_page = assignment[warp_id]*n_elems_per_page;//n_pages_per_warp * warp_id;//assignment[warp_id];
//	if (lane == 0) printf("start_page: %llu\n", (unsigned long long) start_page);
        for (size_t i = 0; i < n_pages_per_warp; i++) {
            //size_t cur_page = start_page + i;
//	    printf("warp_id: %llu\tcur_page: %llu\n", (unsigned long long) warp_id, (unsigned long long) cur_page);
            size_t start_idx = start_page + (i * n_elems_per_sector) + lane;

            for (size_t j = 0; j < n_elems_per_sector; j += 32) {
//		printf("startidx: %llu\n", (unsigned long long) (start_idx+j));
                //if (type == ORIG) {
                    v += ptr[start_idx + j];
                //}
                //else {
                //    v += ptr[start_idx + j];
                //}
            }

        }
        *sum = v;
    }

}*/

template<typename T>
__global__ __launch_bounds__(64,32)
void random_access_warp(array_d_t<T>* dr, uint64_t n_pages_per_warp, unsigned long long* sum, uint64_t type, uint64_t* assignment, uint64_t n_warps, size_t page_size, size_t sector_size) {

    const uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    const uint64_t lane = tid % 32;
    const uint64_t warp_id = tid / 32;
    const uint64_t n_elems_per_page = page_size/sizeof(T);
    const uint64_t n_elems_per_sector = sector_size / sizeof(T);
    T v = 0;
    if (warp_id < n_warps) {
    	bam_ptr<T> ptr(dr);
        size_t start_sector = assignment[warp_id]*n_elems_per_sector;
        size_t start_idx = start_sector + lane;
        for (size_t j = 0; j < n_elems_per_sector; j += 32) {
                    v += ptr[start_idx + j];
        }
        *sum = v;
    }

}

/*
template<typename T>
__global__ __launch_bounds__(64,32)
void random_access_warp(array_d_t<T>* dr, uint64_t n_pages_per_warp, unsigned long long* sum, uint64_t type, uint64_t* assignment, uint64_t n_warps, size_t page_size) {
    const uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    const uint64_t lane = tid % 32;
    const uint64_t warp_id = tid / 32;
    const uint64_t n_elems_per_page = page_size / sizeof(T);
    bam_ptr<T> ptr(dr);
    T v = 0;
    if (warp_id < n_warps) {
        size_t start_page = assignment[warp_id];
	   #pragma nounroll
        for (size_t i = 0; i < n_pages_per_warp; i++) {
            size_t cur_page = start_page + i;
            size_t start_idx = cur_page * n_elems_per_page + lane;
	   #pragma nounroll
            for (size_t j = 0; (start_idx +j) < n_elems_per_page; j += 32) {
                if (type == ORIG) {
                    v += (*dr)[start_idx + j];
                }
                else {
                    v += ptr[start_idx + j];
                }
            }
        }
        *sum = v;
    }
}
*/


int main(int argc, char** argv) {

    Settings settings;
    try
    {
        settings.parseArguments(argc, argv);
    }
    catch (const string& e)
    {
        fprintf(stderr, "%s\n", e.c_str());
        fprintf(stderr, "%s\n", Settings::usageString(argv[0]).c_str());
        return 1;
    }


    hipDeviceProp_t properties;
    if (hipGetDeviceProperties(&properties, settings.cudaDevice) != hipSuccess)
    {
        fprintf(stderr, "Failed to get CUDA device properties\n");
        return 1;
    }

    try {
        //Controller ctrl(settings.controllerPath, settings.nvmNamespace, settings.cudaDevice);

        cuda_err_chk(hipSetDevice(settings.cudaDevice));
        std::vector<Controller*> ctrls(settings.n_ctrls);
        for (size_t i = 0 ; i < settings.n_ctrls; i++)
            ctrls[i] = new Controller(ctrls_paths[i], settings.nvmNamespace, settings.cudaDevice, settings.queueDepth, settings.numQueues);

        //auto dma = createDma(ctrl.ctrl, NVM_PAGE_ALIGN(64*1024*10, 1UL << 16), settings.cudaDevice, settings.adapter, settings.segmentId);

        //std::cout << dma.get()->vaddr << std::endl;
        //QueuePair h_qp(ctrl, settings, 1);
        //std::cout << "in main: " << std::hex << h_qp.sq.cid << "raw: " << h_qp.sq.cid<< std::endl;
        //std::memset(&h_qp, 0, sizeof(QueuePair));
        //prepareQueuePair(h_qp, ctrl, settings, 1);
        //const uint32_t ps, const uint64_t np, const uint64_t c_ps, const Settings& settings, const Controller& ctrl)
        //
        /*
        Controller** d_ctrls;
        cuda_err_chk(hipMalloc(&d_ctrls, n_ctrls*sizeof(Controller*)));
        for (size_t i = 0; i < n_ctrls; i++)
            cuda_err_chk(hipMemcpy(d_ctrls+i, &(ctrls[i]->d_ctrl), sizeof(Controller*), hipMemcpyHostToDevice));
        */
        uint64_t b_size = 64;
        uint64_t g_size = (settings.numThreads + b_size - 1)/b_size;//80*16;
        uint64_t n_threads = b_size * g_size;
        uint64_t n_warps = n_threads/32;


        uint64_t page_size = settings.pageSize;
        uint64_t n_pages = settings.numPages;
        uint64_t total_cache_size = (page_size * n_pages);
        uint64_t sector_size = settings.sectorSize;
        //uint64_t n_pages = total_cache_size/page_size;


        page_cache_t h_pc(page_size, n_pages, sector_size, settings.cudaDevice, ctrls[0][0], (uint64_t) 64, ctrls);
        std::cout << "finished creating cache\n";

        //QueuePair* d_qp;
        page_cache_t* d_pc = (page_cache_t*) (h_pc.d_pc_ptr);
        #define TYPE uint64_t
        uint64_t n_elems = settings.numElems;
        uint64_t t_size = n_elems * sizeof(TYPE);
        uint64_t n_data_pages =  (uint64_t)(t_size/page_size);
        uint64_t n_data_sectors = (uint64_t)(t_size/sector_size);

        range_t<uint64_t> h_range((uint64_t)0, (uint64_t)n_elems, (uint64_t)0, n_data_pages, (uint64_t)0, (uint64_t)page_size, &h_pc, settings.cudaDevice);
        range_t<uint64_t>* d_range = (range_t<uint64_t>*) h_range.d_range_ptr;

        std::vector<range_t<uint64_t>*> vr(1);
        vr[0] = & h_range;
        //(const uint64_t num_elems, const uint64_t disk_start_offset, const std::vector<range_t<T>*>& ranges, Settings& settings)
        array_t<uint64_t> a(n_elems, 0, vr, settings.cudaDevice);


        std::cout << "finished creating range\n";


        uint64_t n_pages_per_warp = settings.numReqs;
        uint64_t gran = settings.gran; //(settings.gran == WARP) ? 32 : b_size;
        uint64_t type = settings.type;

        uint64_t n_elems_per_sector = sector_size / sizeof(uint64_t);
        unsigned long long* d_req_count;
        cuda_err_chk(hipMalloc(&d_req_count, sizeof(unsigned long long)));
        cuda_err_chk(hipMemset(d_req_count, 0, sizeof(unsigned long long)));
        std::cout << "atlaunch kernel\n";
        char st[15];
        cuda_err_chk(hipDeviceGetPCIBusId(st, 15, settings.cudaDevice));
        std::cout << st << std::endl;
        uint64_t* assignment;
        uint64_t* d_assignment;
        if (settings.random) {
            assignment = (uint64_t*) malloc(n_warps*sizeof(uint64_t));
            for (size_t i = 0; i < n_warps; i++) {
                uint64_t sector = rand() % (n_data_sectors);
                assignment[i] = sector;
            }
            cuda_err_chk(hipMalloc(&d_assignment, n_warps*sizeof(uint64_t)));
            cuda_err_chk(hipMemcpy(d_assignment, assignment,  n_warps*sizeof(uint64_t), hipMemcpyHostToDevice));
        }
        Event before;
        //access_kernel<<<g_size, b_size>>>(h_pc.d_ctrls, d_pc, page_size, n_threads, d_req_count, settings.n_ctrls, d_assignment, settings.numReqs);
        if (settings.random) {
                random_access_warp<TYPE><<<g_size, b_size>>>(a.d_array_ptr, n_pages_per_warp, d_req_count, type, d_assignment, n_warps, page_size, sector_size);


        }
        Event after;
        //new_kernel<<<1,1>>>();
        //uint8_t* ret_array = (uint8_t*) malloc(n_pages*page_size);

        //cuda_err_chk(hipMemcpy(ret_array, h_pc.base_addr,page_size*n_pages, hipMemcpyDeviceToHost));
        cuda_err_chk(hipDeviceSynchronize());


        double elapsed = after - before;
        uint64_t ios = n_warps*n_pages_per_warp*n_elems_per_sector;
        uint64_t data = ios*sizeof(uint64_t);
        double iops = ((double)ios)/(elapsed/1000000);
        double bandwidth = (((double)data)/(elapsed/1000000))/(1024ULL*1024ULL*1024ULL);
        a.print_reset_stats();
        std::cout << std::dec << "Elapsed Time: " << elapsed << "\tNumber of Read Ops: "<< ios << "\tData Size (bytes): " << data << std::endl;
        std::cout << std::dec << "Read Ops/sec: " << iops << "\tEffective Bandwidth(GB/S): " << bandwidth << std::endl;

        //std::cout << std::dec << ctrls[0]->ns.lba_data_size << std::endl;

        //std::ofstream ofile("../data", std::ios::binary | std::ios::trunc);
        //ofile.write((char*)ret_array, data);
        //ofile.close();

        for (size_t i = 0 ; i < settings.n_ctrls; i++)
            delete ctrls[i];
        //hexdump(ret_array, n_pages*page_size);
/*
        hipFree(d_qp);
        hipFree(d_pc);
        hipFree(d_req_count);
        free(ret_array);
*/

        //std::cout << "END\n";

        //std::cout << RAND_MAX << std::endl;

    }
    catch (const error& e) {
        fprintf(stderr, "Unexpected error: %s\n", e.what());
        return 1;
    }



}
